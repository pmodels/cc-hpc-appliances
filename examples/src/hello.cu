#include <stdio.h>
#include <mpi.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv) {
    int rank, size;
    int deviceCount;
    char hostname[MPI_MAX_PROCESSOR_NAME];
    int name_len;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Get_processor_name(hostname, &name_len);

    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount == 0) {
        fprintf(stderr,"No CUDA devices found or error occurred: %s\n", hipGetErrorString(err));
        MPI_Finalize();
        return 1;
    }

    int deviceId = rank % deviceCount;
    hipSetDevice(deviceId);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);

    printf("Hello from rank %d of %d on host %s, GPU %d (%s)\n", rank, size, hostname, deviceId, prop.name);

    MPI_Finalize();
    return 0;
}
